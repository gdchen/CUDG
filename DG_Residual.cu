#include "hip/hip_runtime.h"
/* This is the source file for the DG_Residual, containing the function definitions 
 * foe the residual evaluations in DG
 * 
 * Author: Guodong Chen
 * Email: cgderic@umich.edu
 * Last modified: 12/07/2019
 */ 

#include <stdlib.h>
#include "DG_Mesh.cuh"
#include "DG_Quad.cuh"
#include "DG_Basis.cuh"
#include "DG_DataSet.cuh"
#include "DG_All.cuh"
#include "DG_Residual.cuh"
#include "CUDA_Helper.cuh"
#include "DG_Const.cuh"
/* mesh info */
__device__ __constant__ int d_nElem; 
__device__ __constant__ int d_nIFace; 
//
///* basis info */
__device__ __constant__  int d_order;   // p
__device__ __constant__  int d_np;      // np 
__device__ __constant__  int d_nq1;     // nq1 
__device__ __constant__  int d_nq2;     // nq2
__device__ __constant__  double d_sq[MAX_NQ1]; 
__device__ __constant__  double d_wq1[MAX_NQ1];
__device__ __constant__  double d_EdgePhiL[3][MAX_NQ1*MAX_NP];
__device__ __constant__  double d_EdgePhiR[3][MAX_NQ1*MAX_NP];
__device__ __constant__  double d_xyq[2*MAX_NQ2];
__device__ __constant__  double d_wq2[MAX_NQ2];
__device__ __constant__  double d_Phi[MAX_NQ2*MAX_NP];
__device__ __constant__  double d_GPhix[MAX_NQ2*MAX_NP];
__device__ __constant__  double d_GPhiy[MAX_NQ2*MAX_NP]; 
                        
__device__ __constant__  double d_Dwq[MAX_NQ2*MAX_NQ2];
__device__ __constant__  double d_Dwq1[MAX_NQ1*MAX_NQ1];

static hipError_t 
assignConstant(DG_All *All){
  
  int i, j, edge;  
  /* Mesh info */ 
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_nElem),  &(All->Mesh->nElem),  sizeof(int)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_nIFace), &(All->Mesh->nIFace), sizeof(int)));

  /* Basis info */ 
  DG_BasisData *BasisData = All->BasisData; 
  int order = BasisData->order;
  int np    = BasisData->np; 
  int nq1   = BasisData->nq1;
  int nq2   = BasisData->nq2; 
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_order), &(BasisData->order), sizeof(int)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_np),    &(BasisData->np),    sizeof(int)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_nq1),   &(BasisData->nq1),   sizeof(int)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_nq2),   &(BasisData->nq2),   sizeof(int)));
  // pointers 
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_sq),  BasisData->sq,  nq1*sizeof(double)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_wq1), BasisData->wq1, nq1*sizeof(double)));
  for (edge=0; edge<3; edge++){
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgePhiL), BasisData->EdgePhiL[edge], 
                                 nq1*np*sizeof(double), edge*MAX_NQ1*MAX_NP*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgePhiR), BasisData->EdgePhiR[edge],
                                 nq1*np*sizeof(double), edge*MAX_NQ1*MAX_NP*sizeof(double)));
  }
  
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_xyq),   BasisData->xyq,   2*nq2*sizeof(double)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_wq2),   BasisData->wq2,   nq2*sizeof(double)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Phi),   BasisData->Phi,   nq2*np*sizeof(double)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_GPhix), BasisData->GPhix, nq2*np*sizeof(double)));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_GPhiy), BasisData->GPhiy, nq2*np*sizeof(double)));
  
  double *Dwq = (double *) malloc(nq2*nq2*sizeof(double)); 
  for (i=0; i<nq2; i++){
    for (j=0; j<nq2; j++)
      Dwq[i*nq2+j] = 0;
    Dwq[i*nq2+i] = BasisData->wq2[i]; 
  }
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Dwq), Dwq, nq2*nq2*sizeof(double)));
  free(Dwq); 

  double *Dwq1 = (double *)malloc(nq1*nq1*sizeof(double)); 
  for (i=0; i<nq1; i++){
    for (j=0; j<nq1; j++)
      Dwq1[i*nq1+j] = 0;
    Dwq1[i*nq1+i] = BasisData->wq1[i]; 
  }
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Dwq1), Dwq1, nq1*nq1*sizeof(double)));
  free(Dwq1);



  return hipSuccess;
}



// similar to getIntQuadStates duplicate functionality but use const mem
static __device__ hipError_t 
d_getIntQuadStates(double *Uxy, const double *State)
{
  // All of the inputs should be allocated before passed in 
  // Uxy[nq2*NUM_OF_STATES]
  DG_MxM_Set(d_nq2, d_np, NUM_OF_STATES, d_Phi, State, Uxy);
  return hipSuccess;
}

// similar to getEdgeQuadStates duplicate functionlity but use const mem
static __device__ hipError_t 
d_getEdgeQuadStates(double *UL, double *UR, int edge, 
                    const double *StateL, const double *StateR)
{
  DG_MxM_Set(d_nq1, d_np, NUM_OF_STATES, d_EdgePhiL[edge], StateL, UL);
  DG_MxM_Set(d_nq1, d_np, NUM_OF_STATES, d_EdgePhiR[edge], StateR, UR); 
  return hipSuccess;
}


// flux function 
__device__ 
int calculateFlux(double *Fx, double *Fy, int nq2, const double *Uxy)
{
  // All inputs should be allocated before passed in
  // Fx~Fy = [nq2*NUM_OF_STATES], Uxy[nq2*NUM_OF_STATES]
  int i;
  double q1, q2, q3, q4, p;
  for (i=0; i<nq2; i++){
    q1 = Uxy[i*NUM_OF_STATES+0]; 
    q2 = Uxy[i*NUM_OF_STATES+1];
    q3 = Uxy[i*NUM_OF_STATES+2];
    q4 = Uxy[i*NUM_OF_STATES+3];
    p = (GAMMA-1)*(q4-0.5*(q2*q2+q3*q3)/q1); 
    Fx[i*NUM_OF_STATES+0] = q2;
    Fx[i*NUM_OF_STATES+1] = q2*q2/q1+p;
    Fx[i*NUM_OF_STATES+2] = q2*q3/q1;
    Fx[i*NUM_OF_STATES+3] = q2/q1*(q4+p);

    Fy[i*NUM_OF_STATES+0] = q3; 
    Fy[i*NUM_OF_STATES+1] = q2*q3/q1;
    Fy[i*NUM_OF_STATES+2] = q3*q3/q1+p;
    Fy[i*NUM_OF_STATES+3] = q3/q1*(q4+p);

  }  
  return 0;

};



// roe flux 
__device__ 
int getFhat(double *Fhat, const double *UL, const double *UR, const double *normal)
{
  double rL = UL[0]; //printf("%f \n", rL);
  double uL = UL[1]/rL; 
  double vL = UL[2]/rL;
  double unL = uL*normal[0] + vL*normal[1];      //printf("unL: %f \n", unL);
  double qL = sqrt(UL[1]*UL[1]+UL[2]*UL[2])/rL;  //printf("qL: %f \n", qL*rL);
  double pL = (GAMMA-1)*(UL[3]-0.5*rL*qL*qL);    //printf("pL: %f \n", pL);
  if ((pL<=0)||(rL<=0)) {printf("Non-physical 1 state!\n"); return -1;}
  double rHL = UL[3]+pL;
  double HL = rHL/rL;
  //double cL = sqrt(GAMMA*pL/rL);

  //Left flu 
  double FL[NUM_OF_STATES];
  FL[0] = rL*unL;
  FL[1] = UL[1]*unL + pL*normal[0];
  FL[2] = UL[2]*unL + pL*normal[1];
  FL[3] = rHL*unL;

  // process right state
  double rR = UR[0];
  double uR = UR[1]/rR;
  double vR = UR[2]/rR;
  double unR = uR*normal[0] + vR*normal[1];
  double qR = sqrt(UR[1]*UR[1]+UR[2]*UR[2])/rR;
  double pR = (GAMMA-1)*(UR[3]-0.5*rR*qR*qR);
  if ((pR<=0)||(rR<=0)) {printf("Non-physical 2 state! \n"); return -1;}
  double rHR = UR[3] + pR;
  double HR = rHR/rR;
  //double cR = sqrt(GAMMA*pR/rR);

  //right flux 
  double FR[NUM_OF_STATES];
  FR[0] = rR*unR;
  FR[1] = UR[1]*unR + pR*normal[0];
  FR[2] = UR[2]*unR + pR*normal[1];
  FR[3] = rHR*unR;

  // diff in states 
  double du[NUM_OF_STATES];
  int i; 
  for (i=0; i<NUM_OF_STATES; i++) du[i] = UR[i] - UL[i];
  // Roe average 
  double di = sqrt(rR/rL);
  double d1 = 1.0/(1.0+di);
  double ui = (di*uR + uL)*d1;
  double vi = (di*vR + vL)*d1;
  double Hi = (di*HR + HL)*d1; 

  double af = 0.5*(ui*ui+vi*vi);
  double ucp = ui*normal[0] + vi*normal[1];
  double c2 = (GAMMA-1)*(Hi-af);
  if (c2<=0) {printf("Non-physical 3 state! \n"); return -1;}
  double ci = sqrt(c2);
  double ci1 = 1.0/ci;

  double l[3];
  l[0] = ucp + ci;
  l[1] = ucp - ci;
  l[2] = ucp;

  // entropy fix 
  double epsilon = ci*.1;
  for (i=0; i<3; i++){
    if ((l[i]<epsilon) && (l[i]>-epsilon))
      l[i] = 0.5*(epsilon+l[i]*l[i]/epsilon);
    l[i] = fabs(l[i]);
  } 
  double l3 = l[2];

  // average and half-difference of 1st and 2nd eigs 
  double s1 = 0.5*(l[0]+l[1]);
  double s2 = 0.5*(l[0]-l[1]);

  // left eigenvector product generators 
  double G1 = (GAMMA-1)*(af*du[0]-ui*du[1]-vi*du[2] + du[3]);
  double G2 = -ucp*du[0] + du[1]*normal[0] + du[2]*normal[1];

  // requireed functions of G1 and G2
  double C1 = G1*(s1 - l3)*ci1*ci1 + G2*s2*ci1;
  double C2 = G1*s2*ci1            + G2*(s1 - l3);

  Fhat[0] = 0.5*(FL[0]+FR[0]) - 0.5*(l3*du[0] + C1);
  Fhat[1] = 0.5*(FL[1]+FR[1]) - 0.5*(l3*du[1] + C1*ui + C2*normal[0]);
  Fhat[2] = 0.5*(FL[2]+FR[2]) - 0.5*(l3*du[2] + C1*vi + C2*normal[1]);
  Fhat[3] = 0.5*(FL[3]+FR[3]) - 0.5*(l3*du[3] + C1*Hi + C2*ucp);


  double smag = 0;
  for (i=0; i<3; i++) smag = (smag > l[i]) ? smag:l[i]; 
  //printf("smag: %f \n", smag);

  return 0;





}


// calculate roe flux for each edge 
__device__ 
int calculateFhat(double *Fhat, int nq1, const double *UL, const double *UR, const double *normal)
{
  int i;
  for (i=0; i<nq1; i++)
    getFhat(Fhat+i*NUM_OF_STATES, UL+i*NUM_OF_STATES, UR+i*NUM_OF_STATES, normal);
  return 0; 
};




/* kernel to calculate the volume residual */
__global__ void 
calculateVolumeRes(const DG_All *All, double **State, double **R){
  
  int tid = threadIdx.x;   // thread index
  int gid = blockIdx.x*blockDim.x + tid;   // global index
  
  // shared memory 
  int np = d_np; 
  int nq2 = d_nq2; 
  int nElem = d_nElem; 
  // 
  double InvJac[4];  // Inverse Jacobian, only read in once, cant be shared memory 
  int i, j; 
  // global(physical) gradicents, these two array are thread locally dynamic memory 
  double *GPhix = (double *) malloc(nq2*np*sizeof(double)); 
  double *GPhiy = (double *) malloc(nq2*np*sizeof(double)); 
  // get states at interior quad points, thread local dynamic memory  
  double *Uxy = (double *)malloc(nq2*NUM_OF_STATES*sizeof(double)); 
  double *Fx  = (double *)malloc(nq2*NUM_OF_STATES*sizeof(double));
  double *Fy  = (double *)malloc(nq2*NUM_OF_STATES*sizeof(double));
  // temp matrix for flux calculation 
  double *temp = (double *)malloc(np*nq2*sizeof(double)); 
  // all the local dynamically allocated memory do not need initialization
  // as they were set in the function called later
  

  /* each thread calculates the interior volume flux */
  if (gid < nElem){
    //getIntQuadStates(Uxy, State[gid], BasisData); 
    d_getIntQuadStates(Uxy, State[gid]);
    calculateFlux(Fx, Fy, nq2, Uxy);
    // read in once 
    for (i=0; i<4; i++)
      InvJac[i] = All->Mesh->InvJac[gid][i]; 

    for (i=0; i<nq2; i++){
      for (j=0; j<np; j++){
        GPhix[i*np+j] = d_GPhix[i*np+j]*InvJac[0] + d_GPhiy[i*np+j]*InvJac[2];
        GPhiy[i*np+j] = d_GPhix[i*np+j]*InvJac[1] + d_GPhiy[i*np+j]*InvJac[3]; 
      }
    }
    //
    DG_MTxM_Set(np, nq2, nq2, GPhix, d_Dwq, temp); 
    DG_MxM_Set (np, nq2, NUM_OF_STATES, temp, Fx, R[gid]);
    DG_MTxM_Set(np, nq2, nq2, GPhiy, d_Dwq, temp);
    DG_MxM_Add (np, nq2, NUM_OF_STATES, temp, Fy, R[gid]); 
  }

  free(GPhix);  free(GPhiy);  free(Uxy);  free(Fx);  free(Fy); free(temp);

}





/* kernel to calculate the face residual */
__global__ void  
calculateFaceRes(const DG_All *All, double **State, double **RfL, double **RfR){
  
  int tid = threadIdx.x; 
  int gid = blockIdx.x*blockDim.x + tid; 
  int i, j; 
  int np  = d_np; 
  int nq1 = d_nq1; 
  double *UL = (double *)malloc(nq1*NUM_OF_STATES*sizeof(double)); 
  double *UR = (double *)malloc(nq1*NUM_OF_STATES*sizeof(double)); 
  double *Fhat = (double *)malloc(nq1*NUM_OF_STATES*sizeof(double));
  double *temp = (double *)malloc(np*nq1*sizeof(double)); 
  
  int nIFace = d_nIFace; 
  DG_Mesh *Mesh = All->Mesh; 
  int ElemL, ElemR, edge; 
  if (gid < nIFace){
    ElemL = Mesh->IFace[gid].ElemL; 
    ElemR = Mesh->IFace[gid].ElemR; 
    edge  = Mesh->IFace[gid].EdgeL;   // edgeL = edgeR
    //getEdgeQuadStates(UL, UR, edge, State[ElemL], State[ElemR], BasisData); 
    d_getEdgeQuadStates(UL, UR, edge, State[ElemL], State[ElemR]); 
    calculateFhat(Fhat, nq1, UL, UR, Mesh->normal+2*gid);
    
    DG_MTxM_Set(np, nq1, nq1, d_EdgePhiL[edge], d_Dwq1, temp);
    DG_cMxM_Set(Mesh->Length[gid], np, nq1, NUM_OF_STATES, temp, Fhat, RfL[gid]); // sub later

    DG_MTxM_Set(np, nq1, nq1, d_EdgePhiR[edge], d_Dwq1, temp); 
    DG_cMxM_Set(Mesh->Length[gid], np, nq1, NUM_OF_STATES, temp, Fhat, RfR[gid]); // ad later
  }

  free(UL);  free(UR);  free(Fhat);  free(temp); 

}



/* kernel to add volume and face residuals, and convert to RHS for time integration */
__global__ void 
addRes(const DG_All *All, double **R, double **RfL, double **RfR){

  int tid = threadIdx.x; 
  int gid = blockIdx.x * blockDim.x + tid; 
  int i, edge;
  int ndof = d_np * NUM_OF_STATES; 
  //int E2F[3]; 
  //for (edge=0; edge<3; edge++)
  //  E2F[edge] = All->Mesh->E2F[gid]i[edge]; 
  int ElemL, ElemR; 
  int gface; 
  DG_Mesh *Mesh = All->Mesh; 
  if (gid < d_nElem){
    for (edge=0; edge<3; edge++){
      gface = Mesh->E2F[gid][edge]; 
      ElemL = Mesh->IFace[gface].ElemL; 
      ElemR = Mesh->IFace[gface].ElemR; 
      //ElemL = All->Mesh->IFace[All->Mesh->E2F[gid][edge]].ElemL; 
      //ElemR = All->Mesh->IFace[All->Mesh->E2F[gid][edge]].ElemR;
      if (gid == ElemL)
        for (i=0; i<ndof; i++)
          R[gid][i] -= RfL[gface][i];  
      if (gid == ElemR)
        for (i=0; i<ndof; i++)
          R[gid][i] += RfR[gface][i];
    }
  }

}


__global__ void 
getResAllAtOnce(const DG_All *All, double **State, double **R){

}

__global__ void 
Res2RHS(const DG_All *All, double **R, double **f){

  int tid = threadIdx.x; 
  int gid = blockIdx.x * blockDim.x + tid; 
  int np = d_np; 
  if (gid < d_nElem){
    DG_MxM_Set(np, np, NUM_OF_STATES, All->DataSet->InvMassMatrix[gid], R[gid], f[gid]); 
  }

}


/* kernel to update intermediate states in RK4 */
__global__ void 
rk4_inter(DG_All *All, double **State, double dt, double **f){
  int tid = threadIdx.x; 
  int gid = blockIdx.x * blockDim.x + tid; 
  int ndof = d_np * NUM_OF_STATES; 
  int i; 
  if (gid < d_nElem){
    for (i=0; i<ndof; i++)
      State[gid][i] = All->DataSet->State[gid][i] + dt*f[gid][i]; 
  }

}


/* kernel to update the final states in rk4 */
__global__ void
rk4_final(DG_All *All, double dt, double **f0, double **f1, double **f2, double **f3)
{

  int tid = threadIdx.x; 
  int gid = blockIdx.x * blockDim.x + tid; 
  int ndof = d_np * NUM_OF_STATES; 
  int i; 
  if (gid < d_nElem){
    for (i=0; i<ndof; i++)
      All->DataSet->State[gid][i] += dt*(f0[gid][i] + 2*f1[gid][i] + 2*f2[gid][i] + f3[gid][i]);

  }

}


/* host function to lunch kernels performing RK4 time integration */
hipError_t
DG_RK4(DG_All *All, double dt, int Nt){
  
  int nElem = All->Mesh->nElem;  // # of elem 
  int nIFace = All->Mesh->nIFace; // # of faces 
  int np = All->BasisData->np;   // dof per elem 
  int i, j; 
  // temp states, residual, rhs for rk4
  double **U, **R, **f0, **f1, **f2, **f3; 
  double *tempU, *tempR, *tempf0, *tempf1, *tempf2, *tempf3; 
  // memory allocation 
  CUDA_CALL(hipMallocManaged(&U,  nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&R,  nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&f0, nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&f1, nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&f2, nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&f3, nElem*sizeof(double *)));

  CUDA_CALL(hipMallocManaged(&tempU,  nElem*np*NUM_OF_STATES*sizeof(double)));
  CUDA_CALL(hipMallocManaged(&tempR,  nElem*np*NUM_OF_STATES*sizeof(double)));
  CUDA_CALL(hipMallocManaged(&tempf0, nElem*np*NUM_OF_STATES*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&tempf1, nElem*np*NUM_OF_STATES*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&tempf2, nElem*np*NUM_OF_STATES*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&tempf3, nElem*np*NUM_OF_STATES*sizeof(double))); 

  
  double **RfL, **RfR; 
  double *tempRfL, *tempRfR; 
  CUDA_CALL(hipMallocManaged(&RfL, nIFace*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&RfR, nIFace*sizeof(double *)));
  
  CUDA_CALL(hipMallocManaged(&tempRfL, nIFace*np*NUM_OF_STATES*sizeof(double)));
  CUDA_CALL(hipMallocManaged(&tempRfR, nIFace*np*NUM_OF_STATES*sizeof(double)));


  // allocation and initialization 
  for (i=0; i<nElem; i++){
    U[i]  = tempU  + i*np*NUM_OF_STATES; 
    R[i]  = tempR  + i*np*NUM_OF_STATES; 
    f0[i] = tempf0 + i*np*NUM_OF_STATES; 
    f1[i] = tempf1 + i*np*NUM_OF_STATES; 
    f2[i] = tempf2 + i*np*NUM_OF_STATES; 
    f3[i] = tempf3 + i*np*NUM_OF_STATES;  
    for (j=0; j<np*NUM_OF_STATES; j++){
      U[i][j] = 0;
      R[i][j] = 0; 
      f0[i][j] = 0;  f1[i][j] = 0; f2[i][j] = 0; f3[i][j] = 0;
    }
  }

  for (i=0; i<nIFace; i++){
    RfL[i] = tempRfL + i*np*NUM_OF_STATES; 
    RfR[i] = tempRfR + i*np*NUM_OF_STATES;
  }


  int threadPerBlock = 256;
  int elemBlock = (nElem + threadPerBlock - 1)/threadPerBlock; 
  int faceBlock = (nIFace + threadPerBlock -1)/threadPerBlock; 
  


  CUDA_CALL(assignConstant(All));
  printf("elem kernel lunch (%d,%d)\n",elemBlock, threadPerBlock);  
  printf("face kernel lunch (%d,%d)\n",faceBlock, threadPerBlock);  
  // async kernel luncah 
  hipStream_t stream_elem, stream_face; 
  CUDA_CALL(hipStreamCreate(&stream_elem));  
  CUDA_CALL(hipStreamCreate(&stream_face));

  int t = 0; 
  for (t=0; t<Nt; t++){
    // first we need to copy the states data 
    CUDA_CALL(hipMemcpy(U[0], All->DataSet->State[0], nElem*np*NUM_OF_STATES*sizeof(double), 
              hipMemcpyDeviceToDevice)); 
    calculateVolumeRes <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, R);
    calculateFaceRes   <<<faceBlock, threadPerBlock, 0, stream_face>>> (All, U, RfL, RfR); 
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
    addRes             <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, RfL, RfR); 
    Res2RHS            <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, f0); 
    rk4_inter          <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, dt/2, f0);

    calculateVolumeRes <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, R);
    calculateFaceRes   <<<faceBlock, threadPerBlock, 0, stream_face>>> (All, U, RfL, RfR); 
    CUDA_CALL(hipDeviceSynchronize());
    addRes             <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, RfL, RfR); 
    Res2RHS            <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, f1); 
    rk4_inter          <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, dt/2, f1);

    calculateVolumeRes <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, R);
    calculateFaceRes   <<<faceBlock, threadPerBlock, 0, stream_face>>> (All, U, RfL, RfR); 
    CUDA_CALL(hipDeviceSynchronize());
    addRes             <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, RfL, RfR); 
    Res2RHS            <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, f2); 
    rk4_inter          <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, dt, f2);

    calculateVolumeRes <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, U, R);
    calculateFaceRes   <<<faceBlock, threadPerBlock, 0, stream_face>>> (All, U, RfL, RfR); 
    CUDA_CALL(hipDeviceSynchronize());
    addRes             <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, RfL, RfR); 
    Res2RHS            <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, R, f3); 
    rk4_final          <<<elemBlock, threadPerBlock, 0, stream_elem>>> (All, dt/6, f0, f1, f2, f3);
  }
  
  // destory the cudaStream 
  CUDA_CALL(hipStreamDestroy(stream_elem)); 
  CUDA_CALL(hipStreamDestroy(stream_face));
  // free memory 
  CUDA_CALL(hipFree(tempU));  CUDA_CALL(hipFree(U));
  CUDA_CALL(hipFree(tempR));  CUDA_CALL(hipFree(R));
  CUDA_CALL(hipFree(tempf0)); CUDA_CALL(hipFree(f0));
  CUDA_CALL(hipFree(tempf1)); CUDA_CALL(hipFree(f1));
  CUDA_CALL(hipFree(tempf2)); CUDA_CALL(hipFree(f2));
  CUDA_CALL(hipFree(tempf3)); CUDA_CALL(hipFree(f3));
  
  CUDA_CALL(hipFree(tempRfL)); CUDA_CALL(hipFree(RfL));
  CUDA_CALL(hipFree(tempRfR)); CUDA_CALL(hipFree(RfR));

  return hipSuccess;

}
