#include "hip/hip_runtime.h"
/* This file contains the implementation of the DG_DataSet methods 
 *
 * Author: Guodong Chen
 * Email: cgderic@umich.edu 
 * Last modified: 12/05/2019
 */ 

#include "stdlib.h"
#include "DG_Const.cuh"
#include "DG_Mesh.cuh"
#include "DG_Quad.cuh"
#include "DG_Basis.cuh"
#include "DG_Math.cuh"
#include "DG_DataSet.cuh"



hipError_t initDataSet(DG_DataSet *DataSet)
{
  DataSet->nElem = 0;
  DataSet->order = 0;
  DataSet->State = NULL;
  DataSet->MassMatrix = NULL;
  DataSet->InvMassMatrix = NULL; 
  return hipSuccess; 
}





hipError_t createDataSet(DG_DataSet **pDataSet)
{
  CUDA_CALL(hipMallocManaged(pDataSet, sizeof(DG_DataSet))); 
  CUDA_CALL(initDataSet(*pDataSet));
  return hipSuccess; 
}



hipError_t freeDataSet(DG_DataSet *DataSet)
{
  if (DataSet->MassMatrix != NULL){
    CUDA_CALL(hipFree(DataSet->MassMatrix[0])); 
    CUDA_CALL(hipFree(DataSet->InvMassMatrix[0]));
  }
  if (DataSet->State != NULL){
    CUDA_CALL(hipFree(DataSet->State[0]));
  }
  CUDA_CALL(hipFree(DataSet->State));  
  CUDA_CALL(hipFree(DataSet->MassMatrix));  
  CUDA_CALL(hipFree(DataSet->InvMassMatrix));
  CUDA_CALL(hipFree(DataSet));
  return hipSuccess; 
}



hipError_t
computeMassMatrix(DG_DataSet *DataSet, const DG_Mesh *Mesh, const DG_BasisData *BasisData)
{
  int nElem = Mesh->nElem;
  int order = BasisData->order;
  DataSet->nElem = nElem;
  DataSet->order = order; 
  int np = BasisData->np;
  int nq2 = BasisData->nq2; 
  double *Phi = BasisData->Phi; 
  double *wq2 = BasisData->wq2; 
  double *temp; 
  double *tempMass, *tempInvMass; 
  // allocate the memory for the massmatrix and inverse mass matrix 
  CUDA_CALL(hipMallocManaged(&(DataSet->MassMatrix), nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&(DataSet->InvMassMatrix), nElem*sizeof(double *))); 
  CUDA_CALL(hipMallocManaged(&(temp), np*np*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&tempMass, nElem*np*np*sizeof(double)));
  CUDA_CALL(hipMallocManaged(&tempInvMass, nElem*np*np*sizeof(double))); 
  int n, i, j, q2;  
  for (n=0; n<nElem; n++){
    DataSet->MassMatrix[n] = tempMass + n*np*np;
    DataSet->InvMassMatrix[n] = tempInvMass + n*np*np; 
    for (i=0; i<np; i++){
      for (j=0; j<np; j++){
        DataSet->MassMatrix[n][i*np+j] = 0.0;    // Initialization  
        for (q2=0; q2<nq2; q2++){
          DataSet->MassMatrix[n][i*np+j] += Phi[q2*np+i]*Phi[q2*np+j]*wq2[q2]; 
        }
        DataSet->MassMatrix[n][i*np+j] *= Mesh->detJ[n];
        //DataSet->InvMassMatrix[n][i*np+j] = DataSet->MassMatrix[n][i*np+j];
        temp[i*np+j] = DataSet->MassMatrix[n][i*np+j]; 

      }
    }
    // invert the massmatrix to get InvMassMatrix 
    //DG_Inv(np, DataSet->InvMassMatrix[n]); 
    DG_Inv(np, temp, DataSet->InvMassMatrix[n]);
  }
  CUDA_CALL(hipFree(temp));
  return hipSuccess;
}


/* initialize the DataSet using interpolation */
hipError_t interpolateIC(DG_DataSet *DataSet, const DG_Mesh *Mesh)
{
  int nElem = DataSet->nElem;
  int order = DataSet->order; 
  int np = (order+1)*(order+2)/2; 
  int i, j;
  // no need to call hipMalloc as xyGlobal is a temp variable,  
  // only used on host, by value
  double **xyGlobal = (double **)malloc(nElem*np*sizeof(double *));
  for (i=0; i<nElem*np; i++) xyGlobal[i] = (double *)malloc(2*sizeof(double));
  getGlobalLagrangeNodes(order, Mesh, xyGlobal);
  // this chunck of memory is accessiable by both CPU and GPU 
  // use hipMalloc instead 
  double *tempState; 
  CUDA_CALL(hipMallocManaged(&(DataSet->State), nElem*sizeof(double *))); 
  CUDA_CALL(hipMallocManaged(&tempState, nElem*np*NUM_OF_STATES*sizeof(double))); 

  double f0, f1, f2, rho, u, v, p;
  for (i=0; i<nElem; i++){
    DataSet->State[i] = tempState + i*np*NUM_OF_STATES; 
    for (j=0; j<np; j++){
      f0 = getf0(xyGlobal[i*np+j]);
      f1 = getf1(f0);
      f2 = getf2(f0);
      rho = RHO_INF*pow(f1, 1.0/(GAMMA-1));
      u = U_INF - f2*(xyGlobal[i*np+j][1]-X_ORIGINAL[1]);
      v = V_INF + f2*(xyGlobal[i*np+j][0]-X_ORIGINAL[0]);
      p = P_INF*pow(f1, GAMMA/(GAMMA-1));
      DataSet->State[i][j*NUM_OF_STATES+0] = rho;
      DataSet->State[i][j*NUM_OF_STATES+1] = rho*u;
      DataSet->State[i][j*NUM_OF_STATES+2] = rho*v;
      DataSet->State[i][j*NUM_OF_STATES+3] = p/(GAMMA-1) + 0.5*rho*(u*u+v*v);
    }
  }
  for (i=0; i<nElem*np; i++) free(xyGlobal[i]); free(xyGlobal); 
  return hipSuccess; 

}


__device__ __host__ hipError_t 
getIntQuadStates(double *Uxy, const double *State, const DG_BasisData *BasisData)
{
  // All of the inputs should be allocated before passed in 
  // Uxy[nq2*NUM_OF_STATES]
  int np = BasisData->np; 
  int nq2 = BasisData->nq2;
  double *Phi = BasisData->Phi;
  DG_MxM_Set(nq2, np, NUM_OF_STATES, Phi, State, Uxy);
  return hipSuccess;
}

// get states at edge quad points 
__device__ __host__ hipError_t 
getEdgeQuadStates(double *UL, double *UR, int edge, const double *StateL, const double *StateR,
                  const DG_BasisData *BasisData)
{
  int np = BasisData->np;
  int nq1 = BasisData->nq1;
  double **EdgePhiL = BasisData->EdgePhiL;
  double **EdgePhiR = BasisData->EdgePhiR; 
  DG_MxM_Set(nq1, np, NUM_OF_STATES, EdgePhiL[edge], StateL, UL);
  DG_MxM_Set(nq1, np, NUM_OF_STATES, EdgePhiR[edge], StateR, UR); 
  return hipSuccess;
}






// Lagrange Nodes 
hipError_t getLagrangeNodes(int order, double **xy)
{
  // xy should be allocated before passed in, xy[np][2]
  int i,j, counter; 
  // no need to hipMalloc as it's a CPU temp variable, 
  // only used on host by value
  double *x = (double *)malloc((order+1)*sizeof(double)); // 1d Lagrange nodes 
  if (order == 0) x[0] = 0.333333333333333333;
  else {for (i=0; i<order+1; i++) x[i] = 1.0/order*i;} 
  counter = 0;
  for (i=0; i<order+1; i++){
    for (j=0; j<order+1-i; j++){
      xy[counter][0] = x[j];
      xy[counter][1] = x[i];
      counter++;
    }
  }
  free(x); 
  return hipSuccess;
}

// global Lagrange nodes 
hipError_t 
getGlobalLagrangeNodes(int order, const DG_Mesh *Mesh, double **xyGlobal)
{ 
  int np = (order+1)*(order+2)/2;
  int nElem = Mesh->nElem;
  double **coord = Mesh->coord;
  int **E2N = Mesh->E2N;
  double **Jac = Mesh->Jac; 
  int i, j;
  double **xy;
  // no need to call hipMalloc, as it's host temp variable
  xy = (double **)malloc(np*sizeof(double *));
  for (i=0; i<np; i++) xy[i] = (double *)malloc(2*sizeof(double));
  double *x0; 
  getLagrangeNodes(order, xy);  
  for (i=0; i<nElem; i++){
    x0 = coord[E2N[i][0]]; 
    for (j=0; j<np; j++){
      xyGlobal[i*np+j][0] = x0[0] + xy[j][0]*Jac[i][0] + xy[j][1]*Jac[i][1];
      xyGlobal[i*np+j][1] = x0[1] + xy[j][0]*Jac[i][2] + xy[j][1]*Jac[i][3];
    }
  }

  for (i=0; i<np; i++) free(xy[i]); free(xy);
  return hipSuccess;
}


// global quad points 
hipError_t 
getGlobalQuadPoints(double **xyGlobal, const DG_Mesh *Mesh, const DG_BasisData *BasisData)
{
  int nq2 = BasisData->nq2; 
  double *xyq = BasisData->xyq; 
  int nElem = Mesh->nElem;
  double **coord = Mesh->coord;
  int **E2N = Mesh->E2N;
  double **Jac = Mesh->Jac;
  int i,j;
  double *x0, *xy;  
  for (i=0; i<nElem; i++)
  {
    x0 = coord[E2N[i][0]]; 
    for (j=0; j<nq2; j++){
      xy = xyq+2*j; 
      xyGlobal[i*nq2+j][0] = x0[0] + xy[0]*Jac[i][0] + xy[1]*Jac[i][1]; 
      xyGlobal[i*nq2+j][1] = x0[1] + xy[0]*Jac[i][2] + xy[1]*Jac[i][3];  
    }
  }
  return hipSuccess; 
}





/*Functions of exact solution*/
double getf0(double *x)
{
  double abs =   (x[0]-X_ORIGINAL[0])*(x[0]-X_ORIGINAL[0]) 
               + (x[1]-X_ORIGINAL[1])*(x[1]-X_ORIGINAL[1]);
  double f0 = 1.0 - abs/RC/RC; 
  return f0;
}
double getf1(double f0)
{
  double f1;
  f1 = 1.0 -EPSILON*EPSILON*(GAMMA-1.0)*M_INF*M_INF*exp(f0)/8.0/PI/PI;
  return f1; 
}
double getf2(double f0)
{
  double f2; 
  f2 = EPSILON*U_ABS*exp(f0/2.0)/2.0/PI/RC;
  return f2;

}
