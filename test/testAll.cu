#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../DG_All.cuh"
#include "../DG_Mesh.cuh"
#include "../DG_Basis.cuh"
#include "../DG_DataSet.cuh"
#include "../DG_Residual.cuh"
#include "../DG_PostProcess.cuh"




int main(int argc, char const *argv[])
{
  size_t heapSize, a; 
  heapSize = 1.0*1024*1024*1024/8.0 *sizeof(double); 
  CUDA_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize,heapSize)); 
  CUDA_CALL(hipDeviceGetLimit(&a, hipLimitMallocHeapSize));
  printf("Heap size = %ld, %ld\n", heapSize, a);
  int order = 1;
  int N = 200; 
  double halfL = 5;
  DG_All *All;
  createAll(&All);
  getAllFromIC(All, order, N, halfL);
  //int nElem = All->Mesh->nElem;
  //int np = All->BasisData->np;
  //int n, i, j;
  //printf("%d %d %d \n", nElem, All->BasisData->nq1, All->BasisData->nq2);
  double dt = 0.001; 
  int Nt = 10;
//  int Nt = 5414;
//  //int Nt = 14142;
  DG_RK4(All, dt, Nt);
  CUDA_CALL(hipDeviceSynchronize());
  writeStates(All);
//  //writeStatesP(All,4);
//  double eu, es, ep;
//  ErrEst(All, &eu, &es, &ep);
//  printf("%f %f %f \n", sqrt(eu/25), sqrt(es/25), ep/25);
  freeAll(All); 
  CUDA_CALL(hipDeviceReset());
  return 0;
}
