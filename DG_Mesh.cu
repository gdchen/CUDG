#include "hip/hip_runtime.h"
/* This is the source file contains the methods for the Mesh struct 
 *
 * Author: Guodong Chen
 * Email: cgderic@umich.edu 
 * Last modified: 12/04/2019
 */

#include "DG_Mesh.cuh"
#include "CUDA_Helper.cuh"

/* initialize the mesh struct */
hipError_t initMesh(DG_Mesh *Mesh){
 Mesh->halfL = 0;
 Mesh->N = 0;
 Mesh->h = 0;

 Mesh->nNode = 0;     
 Mesh->coord = NULL; // V matrix 

 Mesh->nElem = 0;    
 Mesh->E2N = NULL; // E2N

 Mesh->nIFace = 0;
 Mesh->E2F = NULL;
 Mesh->IFace = NULL;

 Mesh->Jac = NULL;
 Mesh->detJ = NULL;
 Mesh->InvJac = NULL;
 Mesh->Length = NULL;
 Mesh->normal = NULL; 
 return hipSuccess; 

}


/* create Mesh struct: allocate and initialize */
hipError_t createMesh(DG_Mesh **pMesh)
{

  CUDA_CALL(hipMallocManaged(pMesh, sizeof(DG_Mesh))); 
  CUDA_CALL(initMesh(*pMesh));

  return hipSuccess; 
}


/* actually generate the mesh, fill in the mesh struct members */
hipError_t generateMesh(DG_Mesh *Mesh, double halfL, int N)
{
  int i, j;
  int counter; 
  Mesh->halfL = halfL; 
  Mesh->N = N;
  Mesh->h = (2*halfL)/N;
  Mesh->nNode = (N+1)*(N+1);
  double *tempCoord; 
  int *tempE2N; 
  int *tempE2F; 
  // allocate the coords 
  CUDA_CALL(hipMallocManaged(&(Mesh->coord), Mesh->nNode*sizeof(double *)));  
  CUDA_CALL(hipMallocManaged(&(tempCoord), 2*Mesh->nNode*sizeof(double))); 
  for (i=0; i<Mesh->nNode; i++) 
    Mesh->coord[i] = tempCoord + 2*i; 

  // assign coords for every node 
  for (i=0; i<N+1; i++) 
  {
    for (j=0; j<N+1; j++)
    {
      Mesh->coord[i*(N+1)+j][0] = -halfL + j*Mesh->h;
      Mesh->coord[i*(N+1)+j][1] = -halfL + i*Mesh->h;
    }
  }

  Mesh->nElem = 2*N*N;
  // allocate E2N matrix 
  CUDA_CALL(hipMallocManaged(&(Mesh->E2N), Mesh->nElem*sizeof(int *))); 
  CUDA_CALL(hipMallocManaged(&tempE2N, 3*Mesh->nElem*sizeof(int))); 
  for (i=0; i<Mesh->nElem; i++) 
    Mesh->E2N[i] = tempE2N + 3*i; 

  // fill in the E2N matrix 
  counter = 0;
  for (i=0; i<N; i++)
  {
    for (j=0; j<N; j++)
    {
      Mesh->E2N[counter][0] = i*(N+1)+j;
      Mesh->E2N[counter][1] = i*(N+1)+j  +1;
      Mesh->E2N[counter][2] = i*(N+1)+j  +(N+1);
      counter++;
      Mesh->E2N[counter][0] = i*(N+1)+j  +1  +(N+1);
      Mesh->E2N[counter][1] = i*(N+1)+j  +1  +N;  
      Mesh->E2N[counter][2] = i*(N+1)+j  +1; 
      counter++;
    }
  }


  Mesh->nIFace = 3*N*N;

  CUDA_CALL(hipMallocManaged(&(Mesh->E2F), Mesh->nElem*sizeof(int *)));
  CUDA_CALL(hipMallocManaged(&tempE2F,     Mesh->nElem*3*sizeof(int)));
  for (i=0; i<Mesh->nElem; i++)
    Mesh->E2F[i] = &(tempE2F[i*3]); 
  // allocate the interior faces 
  CUDA_CALL(hipMallocManaged(&(Mesh->IFace), Mesh->nIFace*sizeof(DG_IFace))); 

  counter = 0;
  for (i=0; i<N; i++)
  {
    for (j=0; j<N; j++)
    {
      Mesh->IFace[counter].ElemL = i*(2*N)+j*2; 
      Mesh->IFace[counter].ElemR = i*(2*N)+j*2  +1;
      Mesh->IFace[counter].EdgeL = 0;
      Mesh->IFace[counter].EdgeR = 0;
      Mesh->IFace[counter].node[0] = i*(N+1)+j  +1;
      Mesh->IFace[counter].node[1] = i*(N+1)+j  +(N+1);
      Mesh->E2F[Mesh->IFace[counter].ElemL][0] = counter; 
      Mesh->E2F[Mesh->IFace[counter].ElemR][0] = counter; 
      
      counter ++; 
      Mesh->IFace[counter].ElemL = i*(2*N)+j*2;
      if (j==0) Mesh->IFace[counter].ElemR = i*(2*N)+j*2  +2*N-1; // Periodic boundary 
      else  Mesh->IFace[counter].ElemR = i*(2*N)+j*2  -1;
      Mesh->IFace[counter].EdgeL = 1;
      Mesh->IFace[counter].EdgeR = 1;
      Mesh->IFace[counter].node[0] = i*(N+1)+j  +(N+1);
      Mesh->IFace[counter].node[1] = i*(N+1)+j;
      Mesh->E2F[Mesh->IFace[counter].ElemL][1] = counter; 
      Mesh->E2F[Mesh->IFace[counter].ElemR][1] = counter; 

      counter++;
      Mesh->IFace[counter].ElemL = i*(2*N)+j*2;
      if (i==0) Mesh->IFace[counter].ElemR = i*(2*N)+j*2  +(N-1)*(2*N)  +1;  // Periodic boundary 
      else Mesh->IFace[counter].ElemR = i*(2*N)+j*2  -(2*N-1);
      Mesh->IFace[counter].EdgeL = 2;
      Mesh->IFace[counter].EdgeR = 2;
      Mesh->IFace[counter].node[0] = i*(N+1)+j; 
      Mesh->IFace[counter].node[1] = i*(N+1)+j  +1;
      Mesh->E2F[Mesh->IFace[counter].ElemL][2] = counter; 
      Mesh->E2F[Mesh->IFace[counter].ElemR][2] = counter; 

      counter ++;

    }
  }

  return hipSuccess; 

}




/* Compute mesh info, include element joacobian, edge length, edge normal */
hipError_t computeMeshInfo(DG_Mesh *Mesh)
{
  int nElem = Mesh->nElem; 
  int nIFace = Mesh->nIFace;
  DG_IFace *IFace = Mesh->IFace; 
  double **coord = Mesh->coord;
  int **E2N = Mesh->E2N; 
  double *tempJac; 
  double *tempInvJac; 
  // allocate the memory for mesh info 
  CUDA_CALL(hipMallocManaged(&(Mesh->Jac),    nElem*sizeof(double *)));
  CUDA_CALL(hipMallocManaged(&tempJac,        4*nElem*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&(Mesh->detJ),   nElem*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&(Mesh->InvJac), nElem*sizeof(double *))); 
  CUDA_CALL(hipMallocManaged(&tempInvJac,     4*nElem*sizeof(double))); 
  CUDA_CALL(hipMallocManaged(&(Mesh->Length), nIFace*sizeof(double)));
  CUDA_CALL(hipMallocManaged(&(Mesh->normal), nIFace*2*sizeof(double))); 

  int i;
  double *x0, *x1, *x2;  
  for (i=0; i<nElem; i++){
    // allocate Jacobian data 
    Mesh->Jac[i] = tempJac + 4*i; 
    // allocate Inverse Jacbian data 
    Mesh->InvJac[i] = tempInvJac + 4*i; 

    x0 = coord[E2N[i][0]];
    x1 = coord[E2N[i][1]];
    x2 = coord[E2N[i][2]];
    Mesh->Jac[i][0] = x1[0] - x0[0]; 
    Mesh->Jac[i][1] = x2[0] - x0[0];
    Mesh->Jac[i][2] = x1[1] - x0[1]; 
    Mesh->Jac[i][3] = x2[1] - x0[1];
    Mesh->detJ[i] = Mesh->Jac[i][0]*Mesh->Jac[i][3] - Mesh->Jac[i][1]*Mesh->Jac[i][2]; 
    Mesh->InvJac[i][0] = x2[1] - x0[1]; 
    Mesh->InvJac[i][1] = x0[0] - x2[0]; 
    Mesh->InvJac[i][2] = x0[1] - x1[1];
    Mesh->InvJac[i][3] = x1[0] - x0[0]; 
  }

  double xA, yA, xB, yB; 
  for (i=0; i<nIFace; i++){
    xA = coord[IFace[i].node[0]][0];
    yA = coord[IFace[i].node[0]][1]; 
    xB = coord[IFace[i].node[1]][0];
    yB = coord[IFace[i].node[1]][1];
    Mesh->Length[i] = sqrt((xA-xB)*(xA-xB) + (yA-yB)*(yA-yB));
    Mesh->normal[i*2] = (yB-yA)/(Mesh->Length[i]);
    Mesh->normal[i*2+1] = (xA-xB)/(Mesh->Length[i]); 
  }


  return hipSuccess; 

}



/* free the mesh memory */
hipError_t freeMesh(DG_Mesh *Mesh)
{    
  
  // free mesh coord 
  CUDA_CALL(hipFree(Mesh->coord[0]));
  CUDA_CALL(hipFree(Mesh->coord)); 
  
  // free mesh E2N 
  CUDA_CALL(hipFree(Mesh->E2N[0]));
  CUDA_CALL(hipFree(Mesh->E2N));

  // free interior faces 
  CUDA_CALL(hipFree(Mesh->E2F[0]));
  CUDA_CALL(hipFree(Mesh->E2F));
  CUDA_CALL(hipFree(Mesh->IFace));

  // free Jacobian data 
  if (Mesh->Jac != NULL){
    CUDA_CALL(hipFree(Mesh->Jac[0]));
    CUDA_CALL(hipFree(Mesh->InvJac[0]));
    CUDA_CALL(hipFree(Mesh->Jac)); 
    CUDA_CALL(hipFree(Mesh->detJ)); 
    CUDA_CALL(hipFree(Mesh->InvJac));
  }

  // free face length and normal data 
  if (Mesh->Length != NULL) {
    CUDA_CALL(hipFree(Mesh->Length)); 
    CUDA_CALL(hipFree(Mesh->normal));
  }

  CUDA_CALL(hipFree(Mesh));
  return hipSuccess; 

}


